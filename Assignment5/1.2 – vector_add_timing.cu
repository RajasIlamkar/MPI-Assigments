#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

__device__ int A[N];
__device__ int B[N];
__device__ int C[N];

__global__ void vectorAdd() {
    int idx = threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int h_A[N], h_B[N], h_C[N];
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(A), h_A, sizeof(int) * N);
    hipMemcpyToSymbol(HIP_SYMBOL(B), h_B, sizeof(int) * N);

    // Setup CUDA timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vectorAdd<<<1, N>>>();
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    hipMemcpyFromSymbol(h_C, HIP_SYMBOL(C), sizeof(int) * N);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
