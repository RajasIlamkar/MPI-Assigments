#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

__device__ int A[N];
__device__ int B[N];
__device__ int C[N];

__global__ void vectorAdd() {
    int idx = threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int h_A[N], h_B[N], h_C[N];
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(A), h_A, sizeof(int) * N);
    hipMemcpyToSymbol(HIP_SYMBOL(B), h_B, sizeof(int) * N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vectorAdd<<<1, N>>>();
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpyFromSymbol(h_C, HIP_SYMBOL(C), sizeof(int) * N);

    // Measured bandwidth = (Read bytes + Write bytes) / time
    int totalThreads = N;
    size_t readBytes = totalThreads * sizeof(int) * 2;  // A + B
    size_t writeBytes = totalThreads * sizeof(int);     // C

    float seconds = ms / 1000.0f;
    float measuredBW = (readBytes + writeBytes) / (seconds * 1e9);  // GB/s

    std::cout << "Kernel execution time: " << ms << " ms" << std::endl;
    std::cout << "Measured Bandwidth: " << measuredBW << " GB/s" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
